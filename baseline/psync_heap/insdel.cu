#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include "util.hpp"
#include "heap.cuh"

using namespace std;

int main(int argc, char *argv[]) {

    if (argc != 8) {
        cout << "./insert <initNum> <insertNum> <batchSize> <batchNum> <tableSize> <blockNum> <blockSize>\n";
        return -1;
    }

    srand(time(NULL));

    int initNum = atoi(argv[1]) == 0 ?
                    0 : pow(2, atoi(argv[1]));
    int insertNum = atoi(argv[2]) == 0 ?
                    0 : pow(2, atoi(argv[2]));

    int batchSize = atoi(argv[3]);
    int batchNum = atoi(argv[4]);
    int tableSize = atoi(argv[5]);

    int blockNum = atoi(argv[6]);
    int blockSize = atoi(argv[7]);

    initNum = initNum * batchSize;

    struct timeval startTime;
    struct timeval endTime;
    double insertTime = 0;

    printf("init %d insert %d\n", initNum, insertNum);
    // generate <keys, vals> sequence
    int *oriKeys = new int[initNum + insertNum];
    for (int i = 0; i < initNum + insertNum; ++i) {
        oriKeys[i] = rand() % INT_MAX;
//        oriKeys[i] = initNum + insertNum - 1 - i;
    }

    // barrier heap sort
	/* Prepare Heap */
    Heap<int> h_heap(batchNum, batchSize, tableSize);
    Heap<int> *d_heap;
	hipMalloc((void **)&d_heap, sizeof(Heap<int>));
    hipMemcpy(d_heap, &h_heap, sizeof(Heap<int>), hipMemcpyHostToDevice);
	
	/* Prepare data array */
	int *d_Keys;
	hipMalloc((void **)&d_Keys, sizeof(int) * (initNum + insertNum));
    hipMemcpy(d_Keys, oriKeys, sizeof(int) * (initNum + insertNum), hipMemcpyHostToDevice);
	
	/* Prepare ins/del table buffer */
	TB<int> h_insTB(tableSize, batchSize, 0);
	TB<int> *d_insTB;
	hipMalloc((void **)&d_insTB, sizeof(TB<int>));
	hipMemcpy(d_insTB, &h_insTB, sizeof(TB<int>), hipMemcpyHostToDevice);
	
    TB<int> h_delTB(tableSize, batchSize, 1);
	TB<int> *d_delTB;
	hipMalloc((void **)&d_delTB, sizeof(TB<int>));
	hipMemcpy(d_delTB, &h_delTB, sizeof(TB<int>), hipMemcpyHostToDevice);

    int smemSize = 10 * batchSize * sizeof(int);

    bool h_status;
    bool *d_status;
    hipMalloc((void **)&d_status, sizeof(bool));

    // Insert Items
    for (int i = 0; i < initNum / batchSize; i++) {
        insertItems<<<1, blockSize, smemSize>>>(d_heap, 
                                             d_Keys + i * batchSize, 
											 d_insTB);
        hipDeviceSynchronize();

        // Even level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();

        updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();

        // Odd level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();

    }

    while (1) {

        hipMemcpy(&h_status, d_status, sizeof(bool), hipMemcpyDeviceToHost);
        if (h_status) break;

        // Even level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();

        updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();

        hipMemcpy(&h_status, d_status, sizeof(bool), hipMemcpyDeviceToHost);
        if (h_status) break;

        // Odd level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();
    }

    setTime(&startTime);

    // Insert
    for (int i = 0; i < insertNum / batchSize; i++) {
        insertItems<<<1, blockSize, smemSize>>>(d_heap, 
                                                d_Keys + initNum + i * batchSize, 
											    d_insTB);
        hipDeviceSynchronize();

        // Even level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();

        updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();

        // Odd level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();
    }

    while (1) {
        
        hipMemcpy(&h_status, d_status, sizeof(bool), hipMemcpyDeviceToHost);
        if (h_status) break;

        // Even level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();

        updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();

        hipMemcpy(&h_status, d_status, sizeof(bool), hipMemcpyDeviceToHost);
        if (h_status) break;

        // Odd level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();
    }

    for (int i = 0; i < (initNum + insertNum) / batchSize; i++) {
        deleteItems<<<1, blockSize>>>(d_heap, 
									  d_Keys + i * batchSize, 
									  d_insTB,
									  d_delTB);
        hipDeviceSynchronize();

        // Even level delete update
        deleteUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_delTB);
        hipDeviceSynchronize();

        updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();
        // Odd level insert update
        deleteUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_delTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();
    }

    while (1) {

        hipMemcpy(&h_status, d_status, sizeof(bool), hipMemcpyDeviceToHost);
        if (h_status) break;

        // Even level delete update
        deleteUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_delTB);
        hipDeviceSynchronize();

        updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();

        hipMemcpy(&h_status, d_status, sizeof(bool), hipMemcpyDeviceToHost);
        if (h_status) break;

        // Odd level insert update
        deleteUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_delTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB, d_status);
        hipDeviceSynchronize();
    }


    setTime(&endTime);
    insertTime += getTime(&startTime, &endTime);
//    printf("%d %d %d %d %d %.4f\n",
//            batchSize, blockNum, blockSize, initNum, insertNum, insertTime);
    printf("%.4f\n", insertTime);

    hipMemcpy(oriKeys, d_Keys, sizeof(int) * (initNum + insertNum), hipMemcpyDeviceToHost);
    for (int i = 1; i < initNum + insertNum; i++) {
        if (oriKeys[i] < oriKeys[i - 1]) {
            printf("%d %d %d\n", i, oriKeys[i], oriKeys[i - 1]);
            return -1;
        }
        if (oriKeys[i] != i) {
            printf("%d %d\n", i, oriKeys[i]);
            return -1;
        }
    }
    return 0;

}
