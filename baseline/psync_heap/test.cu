#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include "util.hpp"
#include "heap.cuh"

using namespace std;

int main(int argc, char *argv[]) {

    if (argc != 7) {
        cout << "./test <arrayNum> <batchSize> <batchNum> <tableSize> <blockNum> <blockSize>\n";
        return -1;
    }

    srand(time(NULL));

    int arrayNum = atoi(argv[1]);

    int batchSize = atoi(argv[2]);
    int batchNum = atoi(argv[3]);
    int tableSize = atoi(argv[4]);

    int blockNum = atoi(argv[5]);
    int blockSize = atoi(argv[6]);

    struct timeval startTime;
    struct timeval endTime;
    double insertTime = 0, deleteTime = 0;

    // generate <keys, vals> sequence
    int *oriKeys = new int[2 * arrayNum];
    int *oriVals = new int[2 * arrayNum];
    for (int i = 0; i < 2 * arrayNum; ++i) {
        oriKeys[i] = rand() % INT_MAX;
        oriVals[i] = i;
    }

    // barrier heap sort
	/* Prepare Heap */
    Heap<int, int> h_heap(batchNum, batchSize, tableSize);
    Heap<int, int> *d_heap;
	hipMalloc((void **)&d_heap, sizeof(Heap<int, int>));
    hipMemcpy(d_heap, &h_heap, sizeof(Heap<int, int>), hipMemcpyHostToDevice);
	
	/* Prepare data array */
	int *d_Keys, *d_Vals;
	hipMalloc((void **)&d_Keys, sizeof(int) * 2 * arrayNum);
	hipMalloc((void **)&d_Vals, sizeof(int) * 2 * arrayNum);
    hipMemcpy(d_Keys, oriKeys, sizeof(int) * 2 * arrayNum, hipMemcpyHostToDevice);
    hipMemcpy(d_Vals, oriVals, sizeof(int) * 2 * arrayNum, hipMemcpyHostToDevice);
	
	/* Prepare ins/del table buffer */
	TB h_insTB(tableSize, batchSize, 0);
	TB *d_insTB;
	hipMalloc((void **)&d_insTB, sizeof(TB));
	hipMemcpy(d_insTB, &h_insTB, sizeof(TB), hipMemcpyHostToDevice);
	
    TB h_delTB(tableSize, batchSize, 1);
	TB *d_delTB;
	hipMalloc((void **)&d_delTB, sizeof(TB));
	hipMemcpy(d_delTB, &h_delTB, sizeof(TB), hipMemcpyHostToDevice);

    int smemSize = 10 * batchSize * sizeof(int);

    // Insert Items
    for (int i = 0; i < arrayNum / batchSize; i++) {
        insertItems<<<1, blockSize, smemSize>>>(d_heap, 
                                             d_Keys + i * batchSize, 
                                             d_Vals + i * batchSize,
											 d_insTB);
        hipDeviceSynchronize();

        // Even level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();

        updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();

        // Odd level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();
    }

    setTime(&startTime);

    // Insert Items
    for (int i = 0; i < arrayNum / batchSize; i++) {
        insertItems<<<1, blockSize, smemSize>>>(d_heap, 
                                             d_Keys + arrayNum + i * batchSize, 
                                             d_Vals + arrayNum + i * batchSize,
											 d_insTB);
        hipDeviceSynchronize();

        // Even level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();

        updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();

        // Odd level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();
    }

    setTime(&endTime);
    insertTime += getTime(&startTime, &endTime);
    cout << insertTime << " ";
    setTime(&startTime);

    // Delete Items
	for (int i = 0; i < arrayNum / batchSize; i++) {
        deleteItems<<<1, blockSize>>>(d_heap, 
									  d_Keys + i * batchSize, 
									  d_Vals + i * batchSize,
									  d_insTB,
									  d_delTB);
        hipDeviceSynchronize();

        // Even level delete update
        deleteUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_delTB);
        hipDeviceSynchronize();

        updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();
        // Odd level insert update
        deleteUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_delTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();
    }
	
    setTime(&endTime);
    deleteTime = getTime(&startTime, &endTime);
    cout << deleteTime << " " << insertTime + deleteTime << " " << endl;
	
    return 0;

}
