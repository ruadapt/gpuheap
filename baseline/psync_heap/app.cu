#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include "util.hpp"
#include "heap.cuh"

using namespace std;

int main(int argc, char *argv[]) {

    if (argc != 8) {
        cout << "./sort <arrayNum> <numLength> <batchSize> <batchNum> <tableSize> <blockNum> <blockSize>\n";
        return -1;
    }

    srand(time(NULL));

    int arrayNum = atoi(argv[1]);
    int numLength = atoi(argv[2]);

    int batchSize = atoi(argv[3]);
    int batchNum = atoi(argv[4]);
    int tableSize = atoi(argv[5]);

    int blockNum = atoi(argv[6]);
    int blockSize = atoi(argv[7]);

    struct timeval startTime;
    struct timeval endTime;

    // generate <keys, vals> sequence
    int *oriKeys = new int[arrayNum];
    int *oriVals = new int[arrayNum];
    int beginNum = pow(10, numLength);
    for (int i = 0; i < arrayNum; ++i) {
        oriKeys[i] = rand() % (9 * beginNum) + beginNum;
        oriVals[i] = i;
    }

    // thrust sort
    int *thrustKeys;
    int *thrustVals;

    hipMalloc((void **)&thrustKeys, sizeof(int) * arrayNum);
    hipMemcpy(thrustKeys, oriKeys, sizeof(int) * arrayNum, hipMemcpyHostToDevice);
    hipMalloc((void **)&thrustVals, sizeof(int) * arrayNum);
    hipMemcpy(thrustVals, oriVals, sizeof(int) * arrayNum, hipMemcpyHostToDevice);

    thrust::device_ptr<int> key_ptr(thrustKeys);
    thrust::device_ptr<int> val_ptr(thrustVals);
    
    setTime(&startTime);
    
    thrust::sort_by_key(thrust::device, key_ptr, key_ptr + arrayNum, val_ptr);
    hipDeviceSynchronize();

    setTime(&endTime);
    cout << "thrust time: " << getTime(&startTime, &endTime) << "ms\n";

    int *h_tKeys = new int[arrayNum];
    int *h_tVals = new int[arrayNum];

    hipMemcpy(h_tKeys, thrustKeys, sizeof(int) * arrayNum, hipMemcpyDeviceToHost);
    hipMemcpy(h_tVals, thrustVals, sizeof(int) * arrayNum, hipMemcpyDeviceToHost);

    // bitonic heap sort
	/* Prepare Heap */
    Heap<int, int> h_heap(batchNum, batchSize, tableSize);
    Heap<int, int> *d_heap;
	hipMalloc((void **)&d_heap, sizeof(Heap<int, int>));
    hipMemcpy(d_heap, &h_heap, sizeof(Heap<int, int>), hipMemcpyHostToDevice);
	
	/* Prepare data array */
	int *d_Keys, *d_Vals;
	hipMalloc((void **)&d_Keys, sizeof(int) * arrayNum);
	hipMalloc((void **)&d_Vals, sizeof(int) * arrayNum);
    hipMemcpy(d_Keys, oriKeys, sizeof(int) * arrayNum, hipMemcpyHostToDevice);
    hipMemcpy(d_Vals, oriVals, sizeof(int) * arrayNum, hipMemcpyHostToDevice);
	
	/* Prepare ins/del table buffer */
	TB h_insTB(tableSize, batchSize, 0);
	TB *d_insTB;
	hipMalloc((void **)&d_insTB, sizeof(TB));
	hipMemcpy(d_insTB, &h_insTB, sizeof(TB), hipMemcpyHostToDevice);
	
    TB h_delTB(tableSize, batchSize, 1);
	TB *d_delTB;
	hipMalloc((void **)&d_delTB, sizeof(TB));
	hipMemcpy(d_delTB, &h_delTB, sizeof(TB), hipMemcpyHostToDevice);

    int smemSize = 10 * batchSize * sizeof(int);

    setTime(&startTime);

    // Insert Items
    for (int i = 0; i < arrayNum / batchSize; i++) {
        insertItems<<<1, blockSize, smemSize>>>(d_heap, 
                                             d_Keys + i * batchSize, 
                                             d_Vals + i * batchSize,
											 d_insTB);
        hipDeviceSynchronize();

        // Even level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();

        updateTableBuffer<int, int><<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();

#ifdef DEBUG		
 		// Result check
		hipMemcpy(&h_heap, d_heap, sizeof(Heap<int, int>), hipMemcpyDeviceToHost);
		h_heap.printHeap();

		hipMemcpy(&h_insTB, d_insTB, sizeof(TB), hipMemcpyDeviceToHost);
		hipMemcpy(&h_delTB, d_delTB, sizeof(TB), hipMemcpyDeviceToHost);
		h_insTB.printTB();
		h_delTB.printTB();
		cout << "--------------------------------------------" << endl;
#endif

        // Odd level insert update
        insertUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_insTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<int, int><<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();

#ifdef DEBUG		
 		// Result check
		hipMemcpy(&h_heap, d_heap, sizeof(Heap<int, int>), hipMemcpyDeviceToHost);
		h_heap.printHeap();

		hipMemcpy(&h_insTB, d_insTB, sizeof(TB), hipMemcpyDeviceToHost);
		hipMemcpy(&h_delTB, d_delTB, sizeof(TB), hipMemcpyDeviceToHost);
		h_insTB.printTB();
		h_delTB.printTB();
		cout << "--------------------------------------------" << endl;
		cout << "--------------------------------------------" << endl;
#endif
    }

    setTime(&endTime);
    cout << "barrier heap insert time: " << getTime(&startTime, &endTime) << "ms\n";
    setTime(&startTime);

#ifdef DEBUG
    cout << "\n###################################################\n";
#endif
	// Delete Items
	for (int i = 0; i < arrayNum / batchSize; i++) {
        deleteItems<<<1, blockSize>>>(d_heap, 
									  d_Keys + i * batchSize, 
									  d_Vals + i * batchSize,
									  d_insTB,
									  d_delTB);
        hipDeviceSynchronize();

#ifdef DEBUG
        hipMemcpy(oriKeys + i * batchSize, d_Keys + i * batchSize, sizeof(int) * batchSize, hipMemcpyDeviceToHost);
        hipMemcpy(oriVals + i * batchSize, d_Vals + i * batchSize, sizeof(int) * batchSize, hipMemcpyDeviceToHost);
        for (int j = 0; j < batchSize; j++) {
            cout << oriKeys[i * batchSize + j] << " " << oriVals[i * batchSize + j] << " | ";
        }
        cout << endl;
#endif

        // Even level delete update
        deleteUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_delTB);
        hipDeviceSynchronize();

        updateTableBuffer<int, int><<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();
#ifdef DEBUG		
		// Result check
		hipMemcpy(&h_heap, d_heap, sizeof(Heap<int, int>), hipMemcpyDeviceToHost);
		h_heap.printHeap();

		hipMemcpy(&h_insTB, d_insTB, sizeof(TB), hipMemcpyDeviceToHost);
		hipMemcpy(&h_delTB, d_delTB, sizeof(TB), hipMemcpyDeviceToHost);
		h_insTB.printTB();
		h_delTB.printTB();
		cout << "--------------------------------------------" << endl;
#endif
        // Odd level insert update
        deleteUpdate<<<blockNum, blockSize, smemSize>>>(d_heap,
														d_delTB);
        hipDeviceSynchronize();
		
		updateTableBuffer<int, int><<<1, 1>>>(d_heap, d_insTB, d_delTB);
        hipDeviceSynchronize();
#ifdef DEBUG		
		// Result check
		hipMemcpy(&h_heap, d_heap, sizeof(Heap<int, int>), hipMemcpyDeviceToHost);
		h_heap.printHeap();

		hipMemcpy(&h_insTB, d_insTB, sizeof(TB), hipMemcpyDeviceToHost);
		hipMemcpy(&h_delTB, d_delTB, sizeof(TB), hipMemcpyDeviceToHost);
		h_insTB.printTB();
		h_delTB.printTB();
		cout << "--------------------------------------------" << endl;
		cout << "--------------------------------------------" << endl;
#endif
    }
	
    setTime(&endTime);
    cout << "barrier heap delete time: " << getTime(&startTime, &endTime) << "ms\n";

    // Result check
	hipMemcpy(oriKeys, d_Keys, sizeof(int) * arrayNum, hipMemcpyDeviceToHost);
	hipMemcpy(oriVals, d_Vals, sizeof(int) * arrayNum, hipMemcpyDeviceToHost);
#ifdef DEBUG
    for (int i = 0; i < arrayNum; i++) {
		cout << oriKeys[i] << " " << oriVals[i] << " | ";
	}
	cout << endl;
#endif
	for (int i = 0; i < arrayNum; i++) {
		if (oriKeys[i] != h_tKeys[i]) {
            printf("Wrong Answer! id: %d thrust %d heap %d\n", 
                    i, h_tKeys[i], oriKeys[i]);
			return -1;
		}
	}
	cout << "Correct!\n";
	
    return 0;

}
