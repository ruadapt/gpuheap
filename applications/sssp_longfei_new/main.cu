#include <stdio.h>
#include <stdint.h>
#include <time.h>
#include <algorithm>
#include "sssp.cuh"
#include "sssp_config.cuh"

#define VERT(x) ((x) & 0xffffffff)
#define DISTANCE(x) ((x) >> 32)

struct input_line {
  int src, dst;
};

//From StackOverflow
int64_t timespecDiff(struct timespec *timeA_p, struct timespec *timeB_p)
{
  return ((timeA_p->tv_sec * 1000000000) + timeA_p->tv_nsec) -
           ((timeB_p->tv_sec * 1000000000) + timeB_p->tv_nsec);
}

bool sort_input(const input_line &a, const input_line &b) {return a.src < b.src || (a.src == b.src && a.dst < b.dst);}

int main(int argc,char ** argv){
  if(argc != 3){
    printf("Usage: sssp [graph filename] [number of lines]\n");
    return 0;
  }

  int * edge_list_index;
  int * edge_dst;
  int * edge_weight;
  int * distance;

  int vert_count = 0;
  FILE * fin = fopen(argv[1],"r");
  FILE * fout = fopen("output.txt", "w");
  //FILE * ftrace = fopen("trace.txt", "w");
  int input_line_count;
  sscanf(argv[2], " %d", &input_line_count);
  input_line * lines = new input_line[input_line_count * 2];
  for(int i = 0;i < input_line_count;++i){
    fscanf(fin, " %d %d", &(lines[i * 2].src), &(lines[i * 2].dst));
    if(lines[i * 2].src >= vert_count) {vert_count = lines[i * 2].src + 1;}
    if(lines[i * 2].dst >= vert_count) {vert_count = lines[i * 2].dst + 1;}
    lines[i * 2 + 1].src = lines[i * 2].dst;
    lines[i * 2 + 1].dst = lines[i * 2].src;
  }
  std::sort(lines, lines + input_line_count * 2, sort_input);
  int edge_count = input_line_count * 2;

  edge_list_index = new int[vert_count + 1];
  edge_dst = new int[edge_count];
  edge_weight = new int[edge_count];
  distance = new int[vert_count];
  int curr_vert = 0;
  edge_list_index[0] = 0;
  for(int i = 0;i < edge_count;++i){
    while(curr_vert < lines[i].src){++curr_vert; edge_list_index[curr_vert] = i;}
    edge_dst[i] = lines[i].dst;
    edge_weight[i] = 1;
  }
  edge_list_index[vert_count] = edge_count;
  for(int i = 0;i < vert_count;++i){distance[i] = 2147483647;}
  distance[0] = 0;

  int * gpu_edge_list_index, * gpu_edge_dst, * gpu_edge_weight, * gpu_distance;
  hipMalloc((void **)&gpu_edge_list_index, sizeof(int) * (vert_count + 1));
  hipMemcpy(gpu_edge_list_index, edge_list_index, sizeof(int) * (vert_count + 1), hipMemcpyHostToDevice);
  hipMalloc((void **)&gpu_edge_dst, sizeof(int) * edge_count);
  hipMemcpy(gpu_edge_dst, edge_dst, sizeof(int) * edge_count, hipMemcpyHostToDevice);
  hipMalloc((void **)&gpu_edge_weight, sizeof(int) * edge_count);
  hipMemcpy(gpu_edge_weight, edge_weight, sizeof(int) * edge_count, hipMemcpyHostToDevice);
  hipMalloc((void **)&gpu_distance, sizeof(int) * vert_count);
  hipMemcpy(gpu_distance, distance, sizeof(int) * vert_count, hipMemcpyHostToDevice);

  int batch_count = vert_count * 5 / CONFIG_BATCH_SIZE;
  if(batch_count < 3) {batch_count = 3;}
  Heap_With_Aux < unsigned long long, int > cpu_heap(batch_count, CONFIG_BATCH_SIZE, 1ull << 63, 0, CONFIG_THREAD_GROUP_NUM), * gpu_heap;
  hipMalloc((void **)&gpu_heap, sizeof(Heap_With_Aux < unsigned long long, int >));
  hipMemcpy(gpu_heap, &cpu_heap, sizeof(Heap_With_Aux < unsigned long long, int >), hipMemcpyHostToDevice);

  insertInitNode<<<1, 1, 4906>>>(gpu_heap, 0);

  unsigned long long * gpu_inserted_nodes;
  hipMalloc((void **)&gpu_inserted_nodes, sizeof(unsigned long long) * CONFIG_THREAD_GROUP_NUM * CONFIG_BATCH_SIZE * CONFIG_CHUNK_SIZE);
  int * gpu_term_sig;
  hipMalloc((void **)&gpu_term_sig, sizeof(int) * CONFIG_THREAD_GROUP_NUM);

  printf("Preparation complete\n");

  struct timespec start_time, end_time;
  clock_gettime(CLOCK_MONOTONIC, &start_time);

  int iteration = 0;
  do{
    hipMemset(gpu_term_sig, 0, sizeof(int) * CONFIG_THREAD_GROUP_NUM);
    ssspKernel<<<CONFIG_THREAD_GROUP_NUM, CONFIG_THREAD_NUM, 36864>>>(gpu_heap, gpu_edge_list_index, gpu_edge_dst, gpu_edge_weight, gpu_distance, gpu_inserted_nodes, gpu_term_sig);
    ++iteration;
    //if(iteration % 100 == 0) {printf("%d\n", iteration);}
    hipMemcpy(&cpu_heap, gpu_heap, sizeof(Heap_With_Aux < unsigned long long, int >), hipMemcpyDeviceToHost);
  } while(cpu_heap.curr_aux_buf_size > 0 || cpu_heap.heap.itemCount() > 0);

  clock_gettime(CLOCK_MONOTONIC, &end_time);
  printf("Finished in %d iterations\n", iteration);
  int64_t duration = timespecDiff(&end_time, &start_time);
  printf("Microseconds: %ld\n", duration / 1000);

  hipMemcpy(distance, gpu_distance, sizeof(int) * vert_count, hipMemcpyDeviceToHost);
  for(int i = 0;i < vert_count;++i){
    fprintf(fout, "%d %d\n", i, distance[i]);
  }
  return 0;
}
