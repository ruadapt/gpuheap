#include <fstream>
#include <iostream>
#include <vector>
#include <algorithm>

#include "models_pq.cuh"
#include "models_fifo.cuh"
#include "util.cuh"
#include "datastructure.hpp"

using namespace std;

bool cmp(KnapsackItem a, KnapsackItem b)
{
    double r1 = (double)a.first / (double)a.second;
    double r2 = (double)b.first / (double)b.second;
    return r1 > r2;
}

int main(int argc, char *argv[])
{
    if (argc != 11) {
        cout << "./knapsack [dataset] [batchnum] [batchsize] [blocknum] [blocksize] \
            [gcThreshold] [model] [delAllowed] [gcThreshold] [expandThreshold] [endingBlockNum]\n";
        return 1;
    }
    ifstream inputFile;

    int batchNum = atoi(argv[2]);
    int batchSize = atoi(argv[3]);
    int blockNum = atoi(argv[4]);
    int blockSize = atoi(argv[5]);
    int gc_threshold = atoi(argv[6]);
    int model = atoi(argv[7]);
    /*int delAllowed = atoi(argv[7]);*/
    /*int gcThreshold = atoi(argv[8]);*/
    /*int expandThreshold = atoi(argv[9]);*/
    /*int endingBlockNum = atoi(argv[10]);*/

    inputFile.open(argv[1]);

    int capacity, inputSize;
    inputFile >> inputSize >> capacity;

    int *weight = new int[inputSize];
    int *benefit = new int[inputSize];
    float *benefitPerWeight = new float[inputSize];

    for (int i = 0; i < inputSize; i++) {
        inputFile >> benefit[i] >> weight[i];
        benefitPerWeight[i] = (float)benefit[i] / (float)weight[i];
    }

    inputFile.close();

	// Sort items by ppw
	KnapsackItem *items = new KnapsackItem[inputSize];
	for (int i = 0; i < inputSize; i++){
		items[i] = KnapsackItem(benefit[i], weight[i], 0, 0);
	}
	sort(items, items + inputSize, cmp);
	for (int i = 0; i < inputSize; i++){
		benefit[i] = items[i].first;
		weight[i] = items[i].second;
		benefitPerWeight[i] = (float)(benefit[i]) / (float)(weight[i]);

	}
	delete[]items;

	int *d_weight, *d_benefit;
    float *d_benefitPerWeight;
    hipMalloc((void **)&d_weight, sizeof(int) * inputSize);
    hipMalloc((void **)&d_benefit, sizeof(int) * inputSize); 
    hipMalloc((void **)&d_benefitPerWeight, sizeof(float) * inputSize);
    hipMemcpy(d_weight, weight, sizeof(int) * inputSize, hipMemcpyHostToDevice);
    hipMemcpy(d_benefit, benefit, sizeof(int) * inputSize, hipMemcpyHostToDevice);
    hipMemcpy(d_benefitPerWeight, benefitPerWeight, sizeof(float) * inputSize, hipMemcpyHostToDevice);

    int max_benefit = 0;
    int *d_max_benefit;
    hipMalloc((void **)&d_max_benefit, sizeof(int));
    hipMemcpy(d_max_benefit, &max_benefit, sizeof(int), hipMemcpyHostToDevice);

    if (model == 0) /* heap */ {
        oneheap(d_weight, d_benefit, d_benefitPerWeight,
                d_max_benefit, capacity, inputSize,
                batchNum, batchSize, blockNum, blockSize,
                gc_threshold);
    } else if (model == 1) /* fifo queue */ {
        onebuffer(d_weight, d_benefit, d_benefitPerWeight,
                d_max_benefit, capacity, inputSize,
                batchNum, batchSize, blockNum, blockSize,
                gc_threshold);
    }
    hipMemcpy(&max_benefit, d_max_benefit, sizeof(int), hipMemcpyDeviceToHost);

    cout << max_benefit << endl;

    delete[] weight; weight = NULL;
    delete[] benefit; benefit = NULL;
    delete[] benefitPerWeight; benefitPerWeight = NULL;
    hipFree(d_weight);
    hipFree(d_benefit);
    hipFree(d_benefitPerWeight);
    hipFree(d_max_benefit);

    return 0;

}
